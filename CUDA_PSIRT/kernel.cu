#include "hip/hip_runtime.h"
#include "Reconstruction.c"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <windows.h>
#include <time.h>

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

//gambiarra
//#define DEBUG_PRINT

__global__ void run_cuda_psirt(Trajectory* t, Particle* p, int* dev_params, PSIRT* dev_psirt);

void cuda_psirt(PSIRT* host_psirt);

__global__ void run_cuda_psirt(Trajectory* t, Particle* p, int* dev_params, PSIRT* dev_psirt)
{
	// Indice da part�cula a ser tratada nesta thread
	int part_index = blockIdx.x * blockDim.x + threadIdx.x;

	dev_psirt->particles = p;
	dev_psirt->trajectories = t;
	
	dev_psirt->n_projections = dev_params[0];
	dev_psirt->n_trajectories = dev_params[1];
	dev_psirt->n_particles = dev_params[2];

	int is_optimized = 0;
	int is_optimizing_dirty_particle = 0;
	int optim_is_ranked = 0;
	int optim_curr_part = 0;
	int optim_curr_iteration = 0;
	int optim_max_iterations = 100;

	int npart = dev_psirt->n_particles;
	int ttl_trajs = dev_psirt->n_trajectories * dev_psirt->n_projections;

	printf("\r\n===========\r\nSTARTUP CUDA PSIRT\r\n===========\r\nPARAMS:");
	printf("\t(#PROJ)\t(#TRAJ)\t(NPART)\r\n\t%d\t%d\t%d\r\n\r\n",dev_psirt->n_projections, dev_psirt->n_trajectories, dev_psirt->n_particles);

	int done = 0;
	int lim = 0;

	while (!done)
	{
		++lim;
			// ---------------------------
		// *** ATUALIZAR POSICOES DAS PARTICULAS ***
		// ---------------------------
		int i=0,j=0;
		Vector2D resultant_force, resultant_vector;
		//for (i = 0; i < dev_psirt->n_particles; i++) 
		//{
		if (p[part_index].status != DEAD) 
		{		
			set(&resultant_force,0.0,0.0);
			set(&resultant_vector,0.0,0.0);
			for (j = 0; j < ttl_trajs; j++) 
			{
				resultant(&(t[j]),&p[part_index], &resultant_vector);
				sum_void(&resultant_force, &resultant_vector, &resultant_force);
			}
			set(&resultant_force, -resultant_force.x, -resultant_force.y);
			update_particle(&p[part_index], &resultant_force);
		}
		//}																// !!!!!!!!!!!!!!!!!!!!! paralelizar
		__syncthreads();
		// ---------------------------
		// *** CALCULO DE TRAJETORIAS SATISFEITAS ***
		// ---------------------------
		for (i=0;i<dev_psirt->n_particles;i++) dev_psirt->particles[i].current_trajectories = 0; 	// zera #traj de cada particula
		for (i=0;i<ttl_trajs; i++) 
		{
			t[i].n_particulas_atual = 0;
			for (j=0; j<npart; j++)																	// !!!!!!!!!!!!!!!!!!!!! paralelizar
			{
				if (p[j].status == ALIVE)
				{
					float distance_point_line = distance(&p[j].location,&t[i]);
					if (distance_point_line<TRAJ_PART_THRESHOLD)
					{
						t[i].n_particulas_atual++;
						p[j].current_trajectories++;
					}
				}
				
			}
		}
		__syncthreads();
		// ---------------------------
		// *** OTIMIZACAO E CONVERGENCIA ***
		// ---------------------------
		// pre-update otimizacao
		if (is_optimizing_dirty_particle) {
			// OTIMIZANDO
			if (optim_curr_iteration < optim_max_iterations) {
				optim_curr_iteration++;
			}
			// OTIMIZACAO FALHOU (EXCEDEU MAX ITERACOES)
			else {
				//			printf("\r\n[OPTIM]\tPARTICLE #%d LIVED (ITER #%d)",optim_curr_part,optim_curr_iteration);
				p[optim_curr_part].status = ALIVE; // NAO CONSEGUIU REMOVER
				optim_curr_part++;
				is_optimizing_dirty_particle = 0;
			}
		}
		__syncthreads();
		int stable = 0;
		for (i=0;i<ttl_trajs;i++) 
		{
			if (t[i].n_particulas_atual>=t[i].n_particulas_estavel)	stable ++;
		}

		if (lim>5000) {
			
		}

		if (stable==ttl_trajs) // is stable					*************(trecho ok)
		{
			// no optim
			done = 1;


			//if (optim_curr_part < npart) 
			//{
				// optimize
				// ---------------------------
				// *** PRIMEIRO PASSO ***
				// Deve-se ordenar as particulas de acordo
				// com a quantidade de trajetorias a que
				// cada uma atende (0 a 3), ordem crescente.
				// ---------------------------
				/*if (!optim_is_ranked)
				{
					Particle temp;
					for (i = 0; i < npart; i++)
					{
						for (j = npart-1; j > i; j--)
						{
							if ((p[j].current_trajectories < p[i].current_trajectories) & (i!=j))  
							{
								temp = p[i];
								p[i] = p[j];
								p[j] = temp;
							}
						}
					}
					optim_is_ranked = 1;
				}*/

				// ---------------------------
				// CASO ESPECIAL
				// part�cula sem trajetoria, ELIMINAR SEM CHECAR
				// ---------------------------
				//__syncthreads();
			/*	if (p[part_index].current_trajectories == 0)
				{
					is_optimizing_dirty_particle = 0;
					p[part_index].status = DEAD;
					optim_curr_part++;
				}

				// ---------------------------
				// CASO NORMAL
				// part�cula atende a mais de 0 trajet�rias, CHECAR ANTES
				// ---------------------------
				else
				{
					is_optimizing_dirty_particle = 1;

					// COMECAR A CHECAR PARTICULA
					if (p[optim_curr_part].status == ALIVE)
					{
						optim_curr_iteration = 0;
						p[optim_curr_part].status = CHECKING;
					}
					// PARTICULA CHECADA & CONVERGIU -> REMOVER
					else if (p[optim_curr_part].status == CHECKING)
					{
						p[optim_curr_part].status = DEAD;
						optim_curr_part++;
					}
				}*/
			//}
			// (end optim)

			//else done = 1;	// DONE 
		}
	}
}








__global__ void run_cuda_psirt_singlethread(Trajectory* t, Particle* p, int* dev_params, PSIRT* dev_psirt)
{
	dev_psirt->particles = p;
	dev_psirt->trajectories = t;
	
	dev_psirt->n_projections = dev_params[0];
	dev_psirt->n_trajectories = dev_params[1];
	dev_psirt->n_particles = dev_params[2];

	int is_optimized = 0;
	int is_optimizing_dirty_particle = 0;
	int optim_is_ranked = 0;
	int optim_curr_part = 0;
	int optim_curr_iteration = 0;
	int optim_max_iterations = 100;

	int npart = dev_psirt->n_particles;
	int ttl_trajs = dev_psirt->n_trajectories * dev_psirt->n_projections;

	printf("\r\n===========\r\nSTARTUP CUDA PSIRT\r\n===========\r\nPARAMS:");
	printf("\t(#PROJ)\t(#TRAJ)\t(NPART)\r\n\t%d\t%d\t%d\r\n\r\n",dev_psirt->n_projections, dev_psirt->n_trajectories, dev_psirt->n_particles);

	int done = 0;
	int lim = 0;

	while (!done)
	{
		++lim;
			// ---------------------------
		// *** ATUALIZAR POSICOES DAS PARTICULAS ***
		// ---------------------------
		int i=0,j=0;
		Vector2D resultant_force, resultant_vector;
		for (i = 0; i < dev_psirt->n_particles; i++) 
		{
			if (p[i].status != DEAD) 
			{		
				set(&resultant_force,0.0,0.0);
				set(&resultant_vector,0.0,0.0);
				for (j = 0; j < ttl_trajs; j++) 
				{
					resultant(&(t[j]),&p[i], &resultant_vector);
					sum_void(&resultant_force, &resultant_vector, &resultant_force);
				}
				set(&resultant_force, -resultant_force.x, -resultant_force.y);
				update_particle(&p[i], &resultant_force);
			}
		}																// !!!!!!!!!!!!!!!!!!!!! paralelizar
		__syncthreads();
		// ---------------------------
		// *** CALCULO DE TRAJETORIAS SATISFEITAS ***
		// ---------------------------
		for (i=0;i<dev_psirt->n_particles;i++) dev_psirt->particles[i].current_trajectories = 0; 	// zera #traj de cada particula
		for (i=0;i<ttl_trajs; i++) 
		{
			t[i].n_particulas_atual = 0;
			for (j=0; j<npart; j++)																	// !!!!!!!!!!!!!!!!!!!!! paralelizar
			{
				if (p[j].status == ALIVE)
				{
					float distance_point_line = distance(&p[j].location,&t[i]);
					if (distance_point_line<TRAJ_PART_THRESHOLD)
					{
						t[i].n_particulas_atual++;
						p[j].current_trajectories++;
					}
				}
				
			}
		}
		__syncthreads();
		// ---------------------------
		// *** OTIMIZACAO E CONVERGENCIA ***
		// ---------------------------
		// pre-update otimizacao
		if (is_optimizing_dirty_particle) {
			// OTIMIZANDO
			if (optim_curr_iteration < optim_max_iterations) {
				optim_curr_iteration++;
			}
			// OTIMIZACAO FALHOU (EXCEDEU MAX ITERACOES)
			else {
				//			printf("\r\n[OPTIM]\tPARTICLE #%d LIVED (ITER #%d)",optim_curr_part,optim_curr_iteration);
				p[optim_curr_part].status = ALIVE; // NAO CONSEGUIU REMOVER
				optim_curr_part++;
				is_optimizing_dirty_particle = 0;
			}
		}
		__syncthreads();
		int stable = 0;
		for (i=0;i<ttl_trajs;i++) 
		{
			if (t[i].n_particulas_atual>=t[i].n_particulas_estavel)	stable ++;
		}

		if (lim>5000) {
			
		}

		if (stable==ttl_trajs) // is stable					*************(trecho ok)
		{
			// no optim
			done = 1;


			//if (optim_curr_part < npart) 
			//{
				// optimize
				// ---------------------------
				// *** PRIMEIRO PASSO ***
				// Deve-se ordenar as particulas de acordo
				// com a quantidade de trajetorias a que
				// cada uma atende (0 a 3), ordem crescente.
				// ---------------------------
				/*if (!optim_is_ranked)
				{
					Particle temp;
					for (i = 0; i < npart; i++)
					{
						for (j = npart-1; j > i; j--)
						{
							if ((p[j].current_trajectories < p[i].current_trajectories) & (i!=j))  
							{
								temp = p[i];
								p[i] = p[j];
								p[j] = temp;
							}
						}
					}
					optim_is_ranked = 1;
				}*/

				// ---------------------------
				// CASO ESPECIAL
				// part�cula sem trajetoria, ELIMINAR SEM CHECAR
				// ---------------------------
				//__syncthreads();
			/*	if (p[part_index].current_trajectories == 0)
				{
					is_optimizing_dirty_particle = 0;
					p[part_index].status = DEAD;
					optim_curr_part++;
				}

				// ---------------------------
				// CASO NORMAL
				// part�cula atende a mais de 0 trajet�rias, CHECAR ANTES
				// ---------------------------
				else
				{
					is_optimizing_dirty_particle = 1;

					// COMECAR A CHECAR PARTICULA
					if (p[optim_curr_part].status == ALIVE)
					{
						optim_curr_iteration = 0;
						p[optim_curr_part].status = CHECKING;
					}
					// PARTICULA CHECADA & CONVERGIU -> REMOVER
					else if (p[optim_curr_part].status == CHECKING)
					{
						p[optim_curr_part].status = DEAD;
						optim_curr_part++;
					}
				}*/
			//}
			// (end optim)

			//else done = 1;	// DONE 
		}
	}
}









void cuda_psirt(PSIRT* host_psirt)
{
	// 1: COPIAR PROJE��ES/TRAJET�RIAS/PART�CULAS
	
	int n_proj = host_psirt->n_projections;
	int n_traj = host_psirt->n_trajectories;
	int n_ttl_traj = n_proj * n_traj;
	int n_part = host_psirt->n_particles;
	
	Trajectory* traj;
	GPUerrchk(hipMalloc((void**)&traj, sizeof(Trajectory) * n_ttl_traj));
	GPUerrchk(hipMemcpy(traj, host_psirt->trajectories, sizeof(Trajectory) * n_ttl_traj, hipMemcpyHostToDevice));

	Particle *part;
	GPUerrchk(hipMalloc((void**)&part, n_part * sizeof(Particle)));
	GPUerrchk(hipMemcpy(part, host_psirt->particles, sizeof(Particle) * n_part, hipMemcpyHostToDevice));

	// 2. PARAMETROS AUXILIARES
	int params[] = {n_proj, n_traj, n_part};
	int *dev_params;
	GPUerrchk(hipMalloc((void**)&dev_params, sizeof(int)*3));
	GPUerrchk(hipMemcpy(dev_params,params,sizeof(int)*3,hipMemcpyHostToDevice));
	
	// 3. EXECUTAR
	PSIRT* dev_psirt;
	GPUerrchk(hipMalloc((void**)&dev_psirt, sizeof(PSIRT)));


	// (parametros de paraleliza��o)
	int n_elements = host_psirt->n_particles;
	int n_threads_per_block = 32;
	int n_blocks = n_elements/n_threads_per_block;

	hipEvent_t start, start_paralel, stop_1, stop_paralel;
	hipEventCreate(&start);
	hipEventCreate(&start_paralel);
	hipEventCreate(&stop_1);
	hipEventCreate(&stop_paralel);
	
	hipEventRecord(start);
	run_cuda_psirt_singlethread<<<1,1>>>(traj, part, dev_params, dev_psirt);
	hipEventRecord(stop_1);
	
	

	// zerar
	hipDeviceSynchronize();
	GPUerrchk(hipMemcpy(traj, host_psirt->trajectories, sizeof(Trajectory) * n_ttl_traj, hipMemcpyHostToDevice));
	GPUerrchk(hipMemcpy(part, host_psirt->particles, sizeof(Particle) * n_part, hipMemcpyHostToDevice));

	hipEventRecord(start_paralel);
	run_cuda_psirt<<<n_blocks,n_threads_per_block>>>(traj, part, dev_params, dev_psirt);
	hipEventSynchronize(stop_paralel);

	hipDeviceSynchronize();


	float ms_1 = 0, ms_par = 0;
	hipEventElapsedTime(&ms_1, start, stop_1);
	hipEventElapsedTime(&ms_1, start_paralel, stop_paralel);
	hipEventDestroy(start);
	hipEventDestroy(start_paralel);
	hipEventDestroy(stop_1);
	hipEventDestroy(stop_paralel);

	printf ("\r\nFINALIZOU EXEC CUDA (1x1)\r\n TEMPO DE EXECU��O FINAL: %f ms\r\n==============\r\n", ms_1);
	printf ("\r\nFINALIZOU EXEC CUDA (%dx%d)\r\n TEMPO DE EXECU��O FINAL: %f ms\r\n==============\r\n", n_blocks, n_threads_per_block, ms_par);

	// 4. COPIAR DE VOLTA
	Particle *host_plist = host_psirt->particles;
	GPUerrchk(hipMemcpy( host_plist, part, sizeof(Particle) * n_part, hipMemcpyDeviceToHost));
}

int main(int argc, char* argv[])
{
	// Inicializar CUDA
	GPUerrchk(hipSetDevice(0));

	// Preparar par�metros no host
	PSIRT* host_psirt = init_psirt();	

	// Passar par�metros para device, executar & copiar de volta para host
	cuda_psirt(host_psirt);
	
	// Gerar bitmaps
	draw_projection_bitmap(host_psirt);
	draw_reconstruction_bitmap(host_psirt);
	
	// Limpeza & finaliza��o
	free(host_psirt);
	GPUerrchk(hipDeviceReset());

	getchar();

    return 0;
}
