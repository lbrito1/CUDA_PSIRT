#include "hip/hip_runtime.h"
#include "Reconstruction.c"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <windows.h>
#include <time.h>

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

//gambiarra
//#define DEBUG_PRINT

__global__ void run_cuda_psirt(Trajectory* t, Particle* p, int* dev_params, PSIRT* dev_psirt);

void cuda_psirt(PSIRT* host_psirt);

__global__ void run_cuda_psirt(Trajectory* t, Particle* p, int* dev_params, PSIRT* dev_psirt)
{
	dev_psirt->particles = p;
	dev_psirt->trajectories = t;
	
	dev_psirt->n_projections = dev_params[0];
	dev_psirt->n_trajectories = dev_params[1];
	dev_psirt->n_particles = dev_params[2];

	int is_optimized = 0;
	int is_optimizing_dirty_particle = 0;
	int optim_is_ranked = 0;
	int optim_curr_part = 0;
	int optim_curr_iteration = 0;
	int optim_max_iterations = 100;

	int npart = dev_psirt->n_particles;
	int ttl_trajs = dev_psirt->n_trajectories * dev_psirt->n_projections;

	printf("\r\n===========\r\nSTARTUP CUDA PSIRT\r\n===========\r\nPARAMS:");
	printf("\t(#PROJ)\t(#TRAJ)\t(NPART)\r\n\t%d\t%d\t%d\r\n\r\n",dev_psirt->n_projections, dev_psirt->n_trajectories, dev_psirt->n_particles);

	int done = 0;
	int lim = 0;

	while (!done&(++lim<2000))
	{
			// ---------------------------
		// *** ATUALIZAR POSICOES DAS PARTICULAS ***
		// ---------------------------
		update_particles(dev_psirt);

		// ---------------------------
		// *** CALCULO DE TRAJETORIAS SATISFEITAS ***
		// ---------------------------
		int i=0,j=0;
		for (i=0;i<dev_psirt->n_particles;i++) dev_psirt->particles[i].current_trajectories = 0; 	// zera #traj de cada particula
		for (i=0;i<ttl_trajs; i++) 
		{
			t[i].n_particulas_atual = 0;
			for (j=0; j<npart; j++)
			{
				if (p[j].status == ALIVE)
				{
					float distance_point_line = distance(&p[j].location,&t[i]);
					if (distance_point_line<TRAJ_PART_THRESHOLD)
					{
						t[i].n_particulas_atual++;
						p[j].current_trajectories++;
					}
				}
				
			}
		}
		// ---------------------------
		// *** OTIMIZACAO E CONVERGENCIA ***
		// ---------------------------
		// pre-update otimizacao
		if (is_optimizing_dirty_particle) {
			// OTIMIZANDO
			if (optim_curr_iteration < optim_max_iterations) {
				optim_curr_iteration++;
			}
			// OTIMIZACAO FALHOU (EXCEDEU MAX ITERACOES)
			else {
				//			printf("\r\n[OPTIM]\tPARTICLE #%d LIVED (ITER #%d)",optim_curr_part,optim_curr_iteration);
				p[optim_curr_part].status = ALIVE; // NAO CONSEGUIU REMOVER
				optim_curr_part++;
				is_optimizing_dirty_particle = 0;
			}
		}

		int stable = 0;
		for (i=0;i<ttl_trajs;i++) 
		{
			if (t[i].n_particulas_atual>=t[i].n_particulas_estavel)	stable ++;
		}
		if (stable==ttl_trajs) // is stable					*************(trecho ok)
		{
			if (optim_curr_part < npart) 
			{
				// optimize
				// ---------------------------
				// *** PRIMEIRO PASSO ***
				// Deve-se ordenar as particulas de acordo
				// com a quantidade de trajetorias a que
				// cada uma atende (0 a 3), ordem crescente.
				// ---------------------------
				if (!optim_is_ranked)
				{
					Particle temp;
					for (i = 0; i < npart; i++)
					{
						for (j = npart-1; j > i; j--)
						{
							if ((p[j].current_trajectories < p[i].current_trajectories) & (i!=j))  
							{
								temp = p[i];
								p[i] = p[j];
								p[j] = temp;
							}
						}
					}
					optim_is_ranked = 1;
				}

				// ---------------------------
				// CASO ESPECIAL
				// part�cula sem trajetoria, ELIMINAR SEM CHECAR
				// ---------------------------
				if (p[optim_curr_part].current_trajectories == 0)
				{
					is_optimizing_dirty_particle = 0;
					p[optim_curr_part].status = DEAD;
					optim_curr_part++;
				}

				// ---------------------------
				// CASO NORMAL
				// part�cula atende a mais de 0 trajet�rias, CHECAR ANTES
				// ---------------------------
				else
				{
					is_optimizing_dirty_particle = 1;

					// COMECAR A CHECAR PARTICULA
					if (p[optim_curr_part].status == ALIVE)
					{
						optim_curr_iteration = 0;
						p[optim_curr_part].status = CHECKING;
					}
					// PARTICULA CHECADA & CONVERGIU -> REMOVER
					else if (p[optim_curr_part].status == CHECKING)
					{
						p[optim_curr_part].status = DEAD;
						optim_curr_part++;
					}
				}
			}
			// (end optim)

			else done = 1;	// DONE 
		}
	}
}

void cuda_psirt(PSIRT* host_psirt)
{
	// 1: COPIAR PROJE��ES/TRAJET�RIAS/PART�CULAS
	
	int n_proj = host_psirt->n_projections;
	int n_traj = host_psirt->n_trajectories;
	int n_ttl_traj = n_proj * n_traj;
	int n_part = host_psirt->n_particles;
	
	Trajectory* traj;
	GPUerrchk(hipMalloc((void**)&traj, sizeof(Trajectory) * n_ttl_traj));
	GPUerrchk(hipMemcpy(traj, host_psirt->trajectories, sizeof(Trajectory) * n_ttl_traj, hipMemcpyHostToDevice));

	Particle *part;
	GPUerrchk(hipMalloc((void**)&part, n_part * sizeof(Particle)));
	GPUerrchk(hipMemcpy(part, host_psirt->particles, sizeof(Particle) * n_part, hipMemcpyHostToDevice));

	// 2. PARAMETROS AUXILIARES
	int params[] = {n_proj, n_traj, n_part};
	int *dev_params;
	GPUerrchk(hipMalloc((void**)&dev_params, sizeof(int)*3));
	GPUerrchk(hipMemcpy(dev_params,params,sizeof(int)*3,hipMemcpyHostToDevice));
	
	// 3. EXECUTAR
	PSIRT* dev_psirt;
	GPUerrchk(hipMalloc((void**)&dev_psirt, sizeof(PSIRT)));

	run_cuda_psirt<<<1,1>>>(traj, part, dev_params, dev_psirt);

	// 4. COPIAR DE VOLTA
	Particle *host_plist = host_psirt->particles;
	GPUerrchk(hipMemcpy( host_plist, part, sizeof(Particle) * n_part, hipMemcpyDeviceToHost));
}

int main(int argc, char* argv[])
{
	// Inicializar CUDA
	GPUerrchk(hipSetDevice(0));

	// Preparar par�metros no host
	PSIRT* host_psirt = init_psirt();	

	// Passar par�metros para device, executar & copiar de volta para host
	cuda_psirt(host_psirt);
	
	// Gerar bitmaps
	draw_projection_bitmap(host_psirt);
	draw_reconstruction_bitmap(host_psirt);
	
	// Limpeza & finaliza��o
	free(host_psirt);
	GPUerrchk(hipDeviceReset());
    return 0;
}