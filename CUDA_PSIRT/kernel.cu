#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <windows.h>
#include <time.h>
#include <GL/glut.h>

#define RES_X 320
#define RES_Y RES_X

#include <cuda_gl_interop.h>

inline void GPUassert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

GLuint positionsVBO;
struct hipGraphicsResource *positionsVBO_CUDA;

float *dev_x;
float *host_x;

__global__ void CUDA_work(float* x)
{
	if (*x<1.0f) *x += .1;
	else *x = 0.0f;
}

int color = 0;

void opengl_draw()
{
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT);
	
	glBegin(GL_POLYGON);

	glVertex2f(*host_x,-*host_x);
	glVertex2f(0.6,0.6);
	glVertex2f(0.6,0.2);

	glEnd();
	glFlush();	
	Sleep(50);
}
void update()
{
	float h = *host_x;
	GPUerrchk(hipMemcpy(dev_x, host_x, sizeof(float), hipMemcpyHostToDevice));
	CUDA_work<<<1,1>>>(dev_x);
	GPUerrchk(hipMemcpy(host_x, dev_x, sizeof(float), hipMemcpyDeviceToHost));
	h = *host_x;
	printf("\r\n %f",*host_x);
	GPUerrchk(hipDeviceSynchronize());
	glutPostRedisplay();
}

void keyboard_handler (unsigned char key, int x, int y)
{
	if (key == 27) exit(0);	//ESC = exit

}


void init_opengl(int argc, char* argv[])
{
	glutInit(&argc, argv);
	glutCreateWindow("PSIRT_CUDA");
	glutReshapeWindow(RES_X, RES_Y);
	glutDisplayFunc(opengl_draw);
	glutIdleFunc(update);
	glutKeyboardFunc(keyboard_handler);
	glutMainLoop();
}






int main(int argc, char* argv[])
{
	host_x = (float*) malloc(sizeof(float));
	*host_x = 0.0f;
	hipError_t cudaStatus;
	GPUerrchk(hipMalloc((void**)&dev_x, sizeof(float)));
	

	// 3. EXECUTAR / DESENHAR
	init_opengl(argc, argv);

    cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}