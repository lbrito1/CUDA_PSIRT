#include "hip/hip_runtime.h"
/*
 *
 *  Created on: Mar 24, 2013
 *      Author: Leo
 */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <math.h>
#define PI 3.1415926536
#define DEGREE_TO_RAD PI/180.0
#define RAD_TO_DEGREE 180.0/PI

// DATA STRUCTURES
typedef struct { double x,y; } Vector2D;

// HOST FUNCTIONS
Vector2D* new_vector(double x, double y); 
Vector2D* clone(Vector2D* a);

// DEVICE FUNCTIONS
__host__ __device__ void set_vector(Vector2D* a, double x, double y);
__host__ __device__ void sum_void(Vector2D* a, Vector2D* b, Vector2D* result);
__host__ __device__ void minus_void(Vector2D* a, Vector2D* b, Vector2D* result);
__device__ double dot_product(Vector2D* a, Vector2D* b);
__device__ double magnitude(Vector2D* a);
__device__ void mult_constant_void(Vector2D* a, double k);
__device__ void normalize(Vector2D* a);
__host__ __device__ void set(Vector2D* v, double x, double y);
__device__ void normalize_void(Vector2D* a);
__device__ void copyTo(Vector2D* a, Vector2D* b);
__device__ void RotateCounterClockWise(Vector2D* a, float angle);
__device__ void RotateClockWise(Vector2D* a, float angle);
__host__ __device__ double vector_vector_distance(Vector2D* a, Vector2D* b);
__device__ void lerp(Vector2D* a, Vector2D* b, double t, Vector2D* c);
__device__ double distance_point_line(Vector2D* point, Vector2D* line_a, Vector2D* line_b);


__host__ __device__ void set_vector(Vector2D* a, double x, double y)
{
	a->x=x;a->y=y;
}

__host__ __device__ void sum_void(Vector2D* a, Vector2D* b, Vector2D* result)
{
	result->x = a->x+b->x;
	result->y = a->y+b->y;
}

__host__ __device__ void minus_void(Vector2D* a, Vector2D* b, Vector2D* result)
{
	result->x = a->x-b->x;
	result->y = a->y-b->y;
}

__device__ double dot_product(Vector2D* a, Vector2D* b)
{
	return (a->x*b->x + a->y*b->y);
}


__host__ __device__ double magnitude(Vector2D* a)
{
	return sqrt((a->x * a->x)+(a->y * a->y));
}

Vector2D* new_vector(double x, double y) {
	Vector2D* a = (Vector2D*) malloc(sizeof(Vector2D));	//TODO 
	a->x=x;
	a->y=y;
	return a;
}

__host__ __device__ void mult_constant_void(Vector2D* a, double k)
{
	a->x=a->x*k;
	a->y=a->y*k;
}


__host__ __device__ void normalize(Vector2D* a)
{
	mult_constant_void(a,1/magnitude(a));
}

__device__ void normalize_void(Vector2D* a)
{
	double constant = 1/magnitude(a);
	mult_constant_void(a,constant);
}

Vector2D* clone(Vector2D* a)
{
	Vector2D* cloned = (Vector2D*) malloc(sizeof(Vector2D));
	cloned->x = a->x;
	cloned->y = a->y;
	return cloned;
}

__device__ void copyTo(Vector2D* a, Vector2D* b)
{
	b->x = a->x;
	b->y = a->y;
}

__host__ __device__ void RotateCounterClockWise(Vector2D* a, float angle)
{
    double angleRad = angle *DEGREE_TO_RAD;
    double x = a->x * cos(angleRad) - a->y * sin(angleRad);
    double y = a->x * sin(angleRad) + a->y *cos(angleRad);
    a->x = x;
	a->y = y;   
}

__host__ __device__ void set(Vector2D* v, double x, double y)
{
	v->x=x;
	v->y=y;
}


__host__ __device__ void RotateClockWise(Vector2D* a, float angle)
{
	RotateCounterClockWise(a, -angle);
}


__host__ __device__ double vector_vector_distance(Vector2D* a, Vector2D* b)
{
  Vector2D c; 
  minus_void(a,b,&c);
  return magnitude(&c);
}


__host__ __device__ void lerp(Vector2D* a, Vector2D* b, double t, Vector2D* c)
{
	c->x = a->x * t + b->x * (1-t);
	c->y = a->y * t + b->y * (1-t);
	
}

__device__ double distance_point_line(Vector2D* point, Vector2D* line_a, Vector2D* line_b)
{
	double x0,y0,x1,y1,x2,y2;
	x0 = point->x; y0 = point->y;
	x1 = line_a->x; y1 = line_a->y;
	x2 = line_b->x; y2 = line_b->y;

	return ((fabs(((x2-x1)*(y1-y0))-((x1-x0)*(y2-y1)))) / sqrt(pow((x2-x1),2))+pow((y2-y1),2));
}
