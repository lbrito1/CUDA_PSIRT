#include "hip/hip_runtime.h"
#include "Reconstruction.c"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define STATUS_STARTING -1
#define STATUS_RUNNING 0
#define STATUS_CONVERGED 1
#define STATUS_OPTIMIZING 2
#define STATUS_OPTIMIZED 3

#define NPART 64
#define NTRAJ 21


__global__ void ppsirt(Trajectory* t, Particle* p, int* dev_params, PSIRT* dev_psirt, int* iter)
{
	*iter = 0;

	// Indice da part�cula a ser tratada nesta thread
	int part_index = blockIdx.x * blockDim.x + threadIdx.x;

	dev_psirt->particles = p;
	dev_psirt->trajectories = t;
	
	dev_psirt->n_projections = dev_params[0];
	dev_psirt->n_trajectories = dev_params[1];
	dev_psirt->n_particles = dev_params[2];

	int is_optimized = 0;
	int is_optimizing_dirty_particle = 0;
	int optim_is_ranked = 0;
	int optim_curr_part = 0;
	
	int optim_max_iterations = 100;

	Particle sh_p [NPART];
	sh_p[part_index] = p[part_index];
	__syncthreads();

	int npart = dev_psirt->n_particles;
	int ttl_trajs = dev_psirt->n_trajectories * dev_psirt->n_projections;

	int status = STATUS_STARTING;
	int lim = 0;

	double ttl_time_p1 = 0;
	double ttl_time_p2 = 0;

	while (status != STATUS_CONVERGED)
	{
		atomicAdd(&lim, 1);
			// ---------------------------
		// *** ATUALIZAR POSICOES DAS PARTICULAS ***
		// ---------------------------
		int i=0,j=0;
		Vector2D resultant_force, resultant_vector;
		if (sh_p[part_index].status != DEAD) 
		{		
			set(&resultant_force,0.0,0.0);
			set(&resultant_vector,0.0,0.0);
			for (j = 0; j < ttl_trajs; j++) 
			{
				resultant(&(t[j]),&sh_p[part_index], &resultant_vector);
				sum_void(&resultant_force, &resultant_vector, &resultant_force);
			}
			set(&resultant_force, -resultant_force.x, -resultant_force.y);
			update_particle(&sh_p[part_index], &resultant_force);
		}
	
		__syncthreads();
	

		// ---------------------------
		// *** CALCULO DE TRAJETORIAS SATISFEITAS ***
		// ---------------------------
		dev_psirt->particles[part_index].current_trajectories = 0; 	// zera #traj de cada particula
		for (i=0;i<ttl_trajs; i++) 
		{
			t[i].n_particulas_atual = 0;
			float distance_point_line = distance(&sh_p[part_index].location,&t[i]);
			if (distance_point_line<TRAJ_PART_THRESHOLD)
			{
				atomicAdd(&(t[i].n_particulas_atual), 1);
				sh_p[part_index].current_trajectories++;
			}
		}
		
		__syncthreads();
		int stable = 0;
		for (i=0;i<ttl_trajs;i++)  if (t[i].n_particulas_atual>=t[i].n_particulas_estavel)	stable ++;
		

		
		if (stable==ttl_trajs) // is stable					*************(trecho ok)
		{
			status = STATUS_CONVERGED;

		}

		//otimizar
	//	if (status == STATUS_CONVERGED) status = STATUS_OPTIMIZING;
		if (status == STATUS_OPTIMIZING)
		{
		//	int max
		}

		

		
	}

	p[part_index] = sh_p[part_index];
	__syncthreads();

	*iter = lim;
}


